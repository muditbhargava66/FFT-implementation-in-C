#include "hip/hip_runtime.h"
#ifdef __HIPCC__

#include "../../include/fft_gpu.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

/**
 * @file fft_cuda.cu
 * @brief CUDA implementation of GPU-accelerated FFT
 * 
 * New in v2.0.0: Massive performance gains using NVIDIA GPUs
 */

// GPU memory structure
struct fft_gpu_memory {
    void* device_ptr;
    size_t size;
};

// GPU plan structure
struct fft_gpu_plan {
    hipfftHandle cufft_plan;
    int n;
    int batch;
    hipfftType type;
};

// Global CUDA state
static int g_cuda_initialized = 0;
static int g_device_id = 0;

// Error checking macro
#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                __FILE__, __LINE__, hipGetErrorString(error)); \
        return -1; \
    } \
} while(0)

#define CUFFT_CHECK(call) do { \
    hipfftResult error = call; \
    if (error != HIPFFT_SUCCESS) { \
        fprintf(stderr, "cuFFT error at %s:%d: %d\n", \
                __FILE__, __LINE__, error); \
        return -1; \
    } \
} while(0)

// Initialize CUDA
int fft_gpu_init_cuda(void) {
    if (g_cuda_initialized) return 0;
    
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    
    if (device_count == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        return -1;
    }
    
    // Select best device
    int best_device = 0;
    int max_multiprocessors = 0;
    
    for (int i = 0; i < device_count; i++) {
        hipDeviceProp_t props;
        CUDA_CHECK(hipGetDeviceProperties(&props, i));
        
        if (props.multiProcessorCount > max_multiprocessors) {
            max_multiprocessors = props.multiProcessorCount;
            best_device = i;
        }
    }
    
    CUDA_CHECK(hipSetDevice(best_device));
    g_device_id = best_device;
    g_cuda_initialized = 1;
    
    return 0;
}

// Cleanup CUDA
void fft_gpu_cleanup_cuda(void) {
    if (!g_cuda_initialized) return;
    
    hipDeviceReset();
    g_cuda_initialized = 0;
}

// Check availability
int fft_gpu_available_cuda(void) {
    int device_count;
    if (hipGetDeviceCount(&device_count) != hipSuccess) {
        return 0;
    }
    return device_count > 0;
}

// Memory allocation
fft_gpu_memory_t fft_gpu_alloc_cuda(size_t n) {
    fft_gpu_memory_t mem = malloc(sizeof(struct fft_gpu_memory));
    if (!mem) return NULL;
    
    mem->size = n * sizeof(complex_t);
    
    if (hipMalloc(&mem->device_ptr, mem->size) != hipSuccess) {
        free(mem);
        return NULL;
    }
    
    return mem;
}

// Memory free
void fft_gpu_free_cuda(fft_gpu_memory_t mem) {
    if (!mem) return;
    
    hipFree(mem->device_ptr);
    free(mem);
}

// Memory copy host to device
void fft_gpu_copy_h2d_cuda(fft_gpu_memory_t dst, const complex_t* src, size_t n) {
    hipMemcpy(dst->device_ptr, src, n * sizeof(complex_t), 
               hipMemcpyHostToDevice);
}

// Memory copy device to host
void fft_gpu_copy_d2h_cuda(complex_t* dst, fft_gpu_memory_t src, size_t n) {
    hipMemcpy(dst, src->device_ptr, n * sizeof(complex_t),
               hipMemcpyDeviceToHost);
}

// Create FFT plan
fft_gpu_plan_t fft_gpu_plan_1d_cuda(int n, int batch, fft_direction dir) {
    fft_gpu_plan_t plan = malloc(sizeof(struct fft_gpu_plan));
    if (!plan) return NULL;
    
    plan->n = n;
    plan->batch = batch;
    plan->type = (dir == FFT_FORWARD) ? HIPFFT_Z2Z : HIPFFT_Z2Z;
    
    if (batch == 1) {
        if (hipfftPlan1d(&plan->cufft_plan, n, plan->type, 1) != HIPFFT_SUCCESS) {
            free(plan);
            return NULL;
        }
    } else {
        int dims[] = {n};
        if (hipfftPlanMany(&plan->cufft_plan, 1, dims,
                         NULL, 1, n,
                         NULL, 1, n,
                         plan->type, batch) != HIPFFT_SUCCESS) {
            free(plan);
            return NULL;
        }
    }
    
    return plan;
}

// Execute FFT
void fft_gpu_execute_cuda(fft_gpu_plan_t plan, fft_gpu_memory_t in, 
                         fft_gpu_memory_t out, fft_direction dir) {
    hipfftDoubleComplex* in_ptr = (hipfftDoubleComplex*)in->device_ptr;
    hipfftDoubleComplex* out_ptr = (hipfftDoubleComplex*)out->device_ptr;
    
    hipfftExecZ2Z(plan->cufft_plan, in_ptr, out_ptr,
                 (dir == FFT_FORWARD) ? HIPFFT_FORWARD : HIPFFT_BACKWARD);
    
    // Scale for inverse transform
    if (dir == FFT_INVERSE) {
        // Launch scaling kernel
        int threads = 256;
        int blocks = (plan->n * plan->batch + threads - 1) / threads;
        
        // scale_kernel<<<blocks, threads>>>(out_ptr, plan->n * plan->batch, 1.0 / plan->n);
        // TODO: Implement scaling kernel
    }
    
    hipDeviceSynchronize();
}

// Destroy plan
void fft_gpu_destroy_plan_cuda(fft_gpu_plan_t plan) {
    if (!plan) return;
    
    hipfftDestroy(plan->cufft_plan);
    free(plan);
}

// Get device name
const char* fft_gpu_get_device_name_cuda(void) {
    static char name[256];
    hipDeviceProp_t props;
    
    if (hipGetDeviceProperties(&props, g_device_id) == hipSuccess) {
        snprintf(name, sizeof(name), "%s", props.name);
        return name;
    }
    
    return "Unknown CUDA Device";
}

// Get memory info
void fft_gpu_get_memory_info_cuda(size_t* total, size_t* available) {
    hipMemGetInfo(available, total);
}

// Convenience function for 1D FFT
int fft_gpu_dft_1d_cuda(complex_t* in, complex_t* out, int n, fft_direction dir) {
    if (!g_cuda_initialized) {
        if (fft_gpu_init_cuda() != 0) return -1;
    }
    
    // Allocate GPU memory
    fft_gpu_memory_t gpu_in = fft_gpu_alloc_cuda(n);
    fft_gpu_memory_t gpu_out = fft_gpu_alloc_cuda(n);
    
    if (!gpu_in || !gpu_out) {
        fft_gpu_free_cuda(gpu_in);
        fft_gpu_free_cuda(gpu_out);
        return -1;
    }
    
    // Create plan
    fft_gpu_plan_t plan = fft_gpu_plan_1d_cuda(n, 1, dir);
    if (!plan) {
        fft_gpu_free_cuda(gpu_in);
        fft_gpu_free_cuda(gpu_out);
        return -1;
    }
    
    // Copy input to GPU
    fft_gpu_copy_h2d_cuda(gpu_in, in, n);
    
    // Execute FFT
    fft_gpu_execute_cuda(plan, gpu_in, gpu_out, dir);
    
    // Copy result back
    fft_gpu_copy_d2h_cuda(out, gpu_out, n);
    
    // Cleanup
    fft_gpu_destroy_plan_cuda(plan);
    fft_gpu_free_cuda(gpu_in);
    fft_gpu_free_cuda(gpu_out);
    
    return 0;
}

// Custom kernels for optimized operations

__global__ void scale_kernel(hipfftDoubleComplex* data, int n, double scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx].x *= scale;
        data[idx].y *= scale;
    }
}

__global__ void butterfly_kernel(hipfftDoubleComplex* data, int n, int stage) {
    // TODO: Implement custom butterfly kernel for small FFTs
}

#endif // __HIPCC__
